
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void polynomial_expansion (float* poly,int degree,int n,float* array) 
{
  int INX=blockIdx.x*blockDim.x+threadIdx.x;
  if(INX<n)
    {
    float val=0.0;
      float exp=1.0;
      for(int x=0;x<=degree;++x)
      {
        val+=exp*poly[x];
        exp*=array[INX];
      }
      array[INX]=val;
    }
}

int main(int argc, char* argv[]) 
{
    if(argc<3) 
    {
      std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
      return -1;
    }

  int n=atoi(argv[1]); 
  int degree=atoi(argv[2]);
  int nbiter=1;
    float* array=new float[n];
    float* poly=new float[degree+1];
    for(int i=0;i<n;++i)
  {
      array[i]=1.;
  }

    for(int i=0;i<degree+1;++i)
  {
      poly[i]=1.;
  }

    float *DArr,*PArr;

  //start calculating time
    std::chrono::time_point<std::chrono::system_clock> start_time,end_time;
    start_time = std::chrono::system_clock::now();

    hipMalloc(&DArr,n*sizeof(float));
    hipMalloc(&PArr,(degree+1)*sizeof(float));

    hipMemcpy(DArr,array,n*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(PArr,poly,(degree+1)*sizeof(float),hipMemcpyHostToDevice);

    polynomial_expansion<<<(n+255)/256, 256>>>(PArr,degree,n,DArr);
    hipMemcpy(array,DArr,n*sizeof(float),hipMemcpyDeviceToHost);

    hipFree(DArr);
    hipFree(PArr);

    hipDeviceSynchronize();
  {
        bool correct=true;
        int ind;
    for(int i=0;i<n;++i) 
    {
      if(fabs(array[i]-(degree+1))>0.01) 
      {
        correct=false;
        ind=i;
      }
    }
        if(!correct)
        std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  // calculate and print time
    end_time=std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_time=(end_time-start_time)/nbiter;
    std::cout<<n<<" "<<degree<<" "<<elapsed_time.count()<<std::endl;
  
  // free arrays
    delete[] array;
    delete[] poly;

    return 0;
}

